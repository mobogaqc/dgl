#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2019 by Contributors
 * \file geometry/cuda/geometry_op_impl.cc
 * \brief Geometry operator CUDA implementation
 */
#include <dgl/array.h>

#include "../../runtime/cuda/cuda_common.h"
#include "../../c_api_common.h"
#include "../geometry_op.h"

#define THREADS 1024

namespace dgl {
namespace geometry {
namespace impl {

/*!
 * \brief Farthest Point Sampler without the need to compute all pairs of distance.
 * 
 * The input array has shape (N, d), where N is the number of points, and d is the dimension.
 * It consists of a (flatten) batch of point clouds.
 *
 * In each batch, the algorithm starts with the sample index specified by ``start_idx``.
 * Then for each point, we maintain the minimum to-sample distance.
 * Finally, we pick the point with the maximum such distance.
 * This process will be repeated for ``sample_points`` - 1 times.
 */
template <typename FloatType, typename IdType>
__global__ void fps_kernel(const FloatType *array_data, const int64_t batch_size,
                           const int64_t sample_points, const int64_t point_in_batch,
                           const int64_t dim, const IdType *start_idx,
                           FloatType *dist_data, IdType *ret_data) {
  const int64_t thread_idx = threadIdx.x;
  const int64_t batch_idx = blockIdx.x;

  const int64_t array_start = point_in_batch * batch_idx;
  const int64_t ret_start = sample_points * batch_idx;

  __shared__ FloatType dist_max_ht[THREADS];
  __shared__ int64_t dist_argmax_ht[THREADS];

  // start with random initialization
  if (thread_idx == 0) {
    ret_data[ret_start] = (IdType)(start_idx[batch_idx]);
  }

  // sample the rest `sample_points - 1` points
  for (auto i = 0; i < sample_points - 1; i++) {
    __syncthreads();

    // the last sampled point
    int64_t sample_idx = (int64_t)(ret_data[ret_start + i]);
    FloatType dist_max = (FloatType)(-1.);
    int64_t dist_argmax = 0;

    // multi-thread distance calculation
    for (auto j = thread_idx; j < point_in_batch; j += THREADS) {
      FloatType one_dist = (FloatType)(0.);
      for (auto d = 0; d < dim; d++) {
        FloatType tmp = array_data[(array_start + j) * dim + d] -
            array_data[(array_start + sample_idx) * dim + d];
        one_dist += tmp * tmp;
      }

      if (i == 0 || dist_data[array_start + j] > one_dist) {
        dist_data[array_start + j] = one_dist;
      }

      if (dist_data[array_start + j] > dist_max) {
        dist_argmax = j;
        dist_max = dist_data[array_start + j];
      }
    }

    dist_max_ht[thread_idx] = dist_max;
    dist_argmax_ht[thread_idx] = dist_argmax;

    /*
     * \brief Parallel Reduction
     *
     * Suppose the maximum is dist_max_ht[k], where 0 <= k < THREAD.
     * After loop at j = 1, the maximum is propagated to [k-1].
     * After loop at j = 2, the maximum is propagated to the range [k-3] to [k].
     * After loop at j = 4, the maximum is propagated to the range [k-7] to [k].
     * After loop at any j < THREADS, we can see [k - 2*j + 1] to [k] are all covered by the maximum.
     * The max value of j is at least floor(THREAD / 2), and it is sufficient to cover [0] with the maximum.
     */

    for (auto j = 1; j < THREADS; j *= 2) {
      __syncthreads();
      if ((thread_idx + j) < THREADS && dist_max_ht[thread_idx] < dist_max_ht[thread_idx + j]) {
          dist_max_ht[thread_idx] = dist_max_ht[thread_idx + j];
          dist_argmax_ht[thread_idx] = dist_argmax_ht[thread_idx + j];
      }
    }

    if (thread_idx == 0) {
      ret_data[ret_start + i + 1] = (IdType)(dist_argmax_ht[0]);
    }
  }
}

template <DLDeviceType XPU, typename FloatType, typename IdType>
void FarthestPointSampler(NDArray array, int64_t batch_size, int64_t sample_points,
    NDArray dist, IdArray start_idx, IdArray result) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();

  const FloatType* array_data = static_cast<FloatType*>(array->data);

  const int64_t point_in_batch = array->shape[0] / batch_size;
  const int64_t dim = array->shape[1];

  // return value
  IdType* ret_data = static_cast<IdType*>(result->data);

  // distance
  FloatType* dist_data = static_cast<FloatType*>(dist->data);

  // sample for each cloud in the batch
  IdType* start_idx_data = static_cast<IdType*>(start_idx->data);

  fps_kernel<<<batch_size, THREADS, 0, thr_entry->stream>>>(
    array_data, batch_size, sample_points,
    point_in_batch, dim, start_idx_data, dist_data, ret_data);
}

template void FarthestPointSampler<kDLGPU, float, int32_t>(
    NDArray array, int64_t batch_size, int64_t sample_points,
    NDArray dist, IdArray start_idx, IdArray result);
template void FarthestPointSampler<kDLGPU, float, int64_t>(
    NDArray array, int64_t batch_size, int64_t sample_points,
    NDArray dist, IdArray start_idx, IdArray result);
template void FarthestPointSampler<kDLGPU, double, int32_t>(
    NDArray array, int64_t batch_size, int64_t sample_points,
    NDArray dist, IdArray start_idx, IdArray result);
template void FarthestPointSampler<kDLGPU, double, int64_t>(
    NDArray array, int64_t batch_size, int64_t sample_points,
    NDArray dist, IdArray start_idx, IdArray result);

}  // namespace impl
}  // namespace geometry
}  // namespace dgl

#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2019 by Contributors
 * \file array/cuda/array_op_impl.cu
 * \brief Array operator GPU implementation
 */
#include <dgl/array.h>
#include "../../runtime/cuda/cuda_common.h"

namespace dgl {
using runtime::NDArray;
namespace aten {
namespace impl {

int FindNumThreads(int dim, int max_nthrs) {
  int ret = max_nthrs;
  while (ret > dim) {
    ret = ret >> 1;
  }
  return ret;
}


///////////////////////////// Range /////////////////////////////

template <typename IdType>
__global__ void _RangeKernel(IdType* out, IdType low, IdType length) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = low + tx;
    tx += stride_x;
  }
}

template <DLDeviceType XPU, typename IdType>
IdArray Range(IdType low, IdType high, DLContext ctx) {
  CHECK(high >= low) << "high must be bigger than low";
  const IdType length = high - low;
  IdArray ret = NewIdArray(length, ctx, sizeof(IdType) * 8);
  if (length == 0)
    return ret;
  IdType* ret_data = static_cast<IdType*>(ret->data);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  int nt = FindNumThreads(length, 1024);
  int nb = (length + nt - 1) / nt;
  _RangeKernel<IdType><<<nb, nt, 0, thr_entry->stream>>>(ret_data, low, length);
  return ret;
}

template IdArray Range<kDLGPU, int32_t>(int32_t, int32_t, DLContext);
template IdArray Range<kDLGPU, int64_t>(int64_t, int64_t, DLContext);

///////////////////////////// AsNumBits /////////////////////////////

template <typename InType, typename OutType>
__global__ void _CastKernel(const InType* in, OutType* out, size_t length) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[tx] = in[tx];
    tx += stride_x;
  }
}

template <DLDeviceType XPU, typename IdType>
IdArray AsNumBits(IdArray arr, uint8_t bits) {
  const std::vector<int64_t> shape(arr->shape, arr->shape + arr->ndim);
  IdArray ret = IdArray::Empty(shape, DLDataType{kDLInt, bits, 1}, arr->ctx);
  const int64_t length = ret.NumElements();
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  int nt = FindNumThreads(length, 1024);
  int nb = (length + nt - 1) / nt;
  if (bits == 32) {
    _CastKernel<IdType, int32_t><<<nb, nt, 0, thr_entry->stream>>>(
        static_cast<IdType*>(arr->data), static_cast<int32_t*>(ret->data), length);
  } else {
    _CastKernel<IdType, int64_t><<<nb, nt, 0, thr_entry->stream>>>(
        static_cast<IdType*>(arr->data), static_cast<int64_t*>(ret->data), length);
  }
  return ret;
}


template IdArray AsNumBits<kDLGPU, int32_t>(IdArray arr, uint8_t bits);
template IdArray AsNumBits<kDLGPU, int64_t>(IdArray arr, uint8_t bits);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
